#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <list>
#define SIGN(x) (x > 1) ? (short)1 : (short)0
#define NUM_STATES (1<<18)//23 maior que 17 da erro no vetor h_transients
#define NUM_COPYS  (1<<20)//22  10
#define NUM_NOS 96
#define N 3

using namespace std;
//typedef unsigned long uint64;
typedef unsigned int uint32;

__device__
bool comp(uint32 *S0, uint32 *S1){
    for (int i = 0; i < N; ++i) {
        if(S1[i] != S0[i])
            return false;
    }
    return true;
}

__device__
short getDecValue(short v){
    return v? v:(short)-1;
}

__device__
short getBit(int idx,uint32 v){
    idx = idx % 32;
    return (short)((v >> idx) & 1);
}

__device__
void setBit(int idx, short newV, uint32 *v){
    idx = idx % 32;
    *v &= ~(1 << idx);
    *v |= ((uint32)newV << idx);
}

__device__
short getBlockIdx(int idx){
    if(idx<32)
        return 0;
    else if(idx>63)
        return 2;
    else
        return 1;
}

__device__
void pass(uint32 *S) {
    uint32 Sc[N];
    for (int i = 0; i < N; ++i) {
        Sc[i] = S[i];
    }
    setBit(0,1,&S[getBlockIdx(0)]);
    setBit(1,1,&S[getBlockIdx(1)]);
    setBit(2,0,&S[getBlockIdx(2)]);
    setBit(3,0,&S[getBlockIdx(3)]);
    setBit(4,0,&S[getBlockIdx(4)]);
    setBit(5,0,&S[getBlockIdx(5)]);
    setBit(6,1,&S[getBlockIdx(6)]);
    setBit(7,1,&S[getBlockIdx(7)]);
    setBit(8,SIGN( + getDecValue(getBit(34,Sc[getBlockIdx(34)]))),&S[getBlockIdx(8)]);
    setBit(9,SIGN( + getDecValue(getBit(0,Sc[getBlockIdx(0)])) + getDecValue(getBit(71,Sc[getBlockIdx(71)]))),&S[getBlockIdx(9)]);
    setBit(10,SIGN( + getDecValue(getBit(43,Sc[getBlockIdx(43)])) + getDecValue(getBit(60,Sc[getBlockIdx(60)])) + (short)(-1)),&S[getBlockIdx(10)]);
    setBit(11,SIGN( - (getDecValue(getBit(2,Sc[getBlockIdx(2)]))) + (short) 1),&S[getBlockIdx(11)]);
    setBit(12,SIGN( - (getDecValue(getBit(13,Sc[getBlockIdx(13)]))) + (short) 1),&S[getBlockIdx(12)]);
    setBit(13,SIGN( + getDecValue(getBit(14,Sc[getBlockIdx(14)])) + getDecValue(getBit(26,Sc[getBlockIdx(26)]))),&S[getBlockIdx(13)]);
    setBit(14,SIGN( + getDecValue(getBit(1,Sc[getBlockIdx(1)]))),&S[getBlockIdx(14)]);
    setBit(15,SIGN( + getDecValue(getBit(2,Sc[getBlockIdx(2)])) - (getDecValue(getBit(4,Sc[getBlockIdx(4)])))),&S[getBlockIdx(15)]);
    setBit(16,SIGN( - (getDecValue(getBit(12,Sc[getBlockIdx(12)]))) + getDecValue(getBit(14,Sc[getBlockIdx(14)])) + (short) 1),&S[getBlockIdx(16)]);
    setBit(17,SIGN( + getDecValue(getBit(16,Sc[getBlockIdx(16)])) + getDecValue(getBit(78,Sc[getBlockIdx(78)]))),&S[getBlockIdx(17)]);
    setBit(18,SIGN( + getDecValue(getBit(17,Sc[getBlockIdx(17)])) - (getDecValue(getBit(7,Sc[getBlockIdx(7)]))) - (getDecValue(getBit(92,Sc[getBlockIdx(92)]))) + (short) 1),&S[getBlockIdx(18)]);
    setBit(19,SIGN( + getDecValue(getBit(18,Sc[getBlockIdx(18)])) + getDecValue(getBit(34,Sc[getBlockIdx(34)])) + getDecValue(getBit(38,Sc[getBlockIdx(38)]))),&S[getBlockIdx(19)]);
    setBit(20,SIGN( + getDecValue(getBit(13,Sc[getBlockIdx(13)])) + getDecValue(getBit(25,Sc[getBlockIdx(25)])) + getDecValue(getBit(33,Sc[getBlockIdx(33)])) - (getDecValue(getBit(37,Sc[getBlockIdx(37)]))) - (getDecValue(getBit(43,Sc[getBlockIdx(43)]))) + getDecValue(getBit(82,Sc[getBlockIdx(82)]))),&S[getBlockIdx(20)]);
    setBit(21,SIGN( + getDecValue(getBit(18,Sc[getBlockIdx(18)])) + getDecValue(getBit(20,Sc[getBlockIdx(20)])) + getDecValue(getBit(20,Sc[getBlockIdx(20)])) - (getDecValue(getBit(76,Sc[getBlockIdx(76)]))) + getDecValue(getBit(95,Sc[getBlockIdx(95)])) + (short)(-1)),&S[getBlockIdx(21)]);
    setBit(22,SIGN( + getDecValue(getBit(13,Sc[getBlockIdx(13)])) + getDecValue(getBit(16,Sc[getBlockIdx(16)]))),&S[getBlockIdx(22)]);
    setBit(23,SIGN( + getDecValue(getBit(22,Sc[getBlockIdx(22)]))),&S[getBlockIdx(23)]);
    setBit(24,SIGN( + getDecValue(getBit(19,Sc[getBlockIdx(19)])) + getDecValue(getBit(23,Sc[getBlockIdx(23)]))),&S[getBlockIdx(24)]);
    setBit(25,SIGN( + getDecValue(getBit(18,Sc[getBlockIdx(18)])) + getDecValue(getBit(19,Sc[getBlockIdx(19)])) + (short)(-1)),&S[getBlockIdx(25)]);
    setBit(26,SIGN( - (getDecValue(getBit(43,Sc[getBlockIdx(43)]))) + getDecValue(getBit(5,Sc[getBlockIdx(5)]))),&S[getBlockIdx(26)]);
    setBit(27,SIGN( + getDecValue(getBit(26,Sc[getBlockIdx(26)]))),&S[getBlockIdx(27)]);
    setBit(28,SIGN( + getDecValue(getBit(7,Sc[getBlockIdx(7)])) + (short) 1),&S[getBlockIdx(28)]);
    setBit(29,SIGN( - (getDecValue(getBit(24,Sc[getBlockIdx(24)]))) - (getDecValue(getBit(25,Sc[getBlockIdx(25)]))) - (getDecValue(getBit(27,Sc[getBlockIdx(27)]))) - (getDecValue(getBit(33,Sc[getBlockIdx(33)]))) + (short)3),&S[getBlockIdx(29)]);
    setBit(30,SIGN( + getDecValue(getBit(28,Sc[getBlockIdx(28)])) + getDecValue(getBit(29,Sc[getBlockIdx(29)])) + (short)(-1)),&S[getBlockIdx(30)]);
    setBit(31,SIGN( - (getDecValue(getBit(30,Sc[getBlockIdx(30)]))) - (getDecValue(getBit(43,Sc[getBlockIdx(43)]))) + (short) 1),&S[getBlockIdx(31)]);
    setBit(32,SIGN( - (getDecValue(getBit(10,Sc[getBlockIdx(10)]))) + getDecValue(getBit(21,Sc[getBlockIdx(21)])) + getDecValue(getBit(84,Sc[getBlockIdx(84)]))),&S[getBlockIdx(32)]);
    setBit(33,SIGN( + getDecValue(getBit(15,Sc[getBlockIdx(15)])) + getDecValue(getBit(25,Sc[getBlockIdx(25)])) + getDecValue(getBit(42,Sc[getBlockIdx(42)])) - (getDecValue(getBit(72,Sc[getBlockIdx(72)]))) + (short)(-1)),&S[getBlockIdx(33)]);
    setBit(34,SIGN( + getDecValue(getBit(4,Sc[getBlockIdx(4)])) + getDecValue(getBit(33,Sc[getBlockIdx(33)])) - (getDecValue(getBit(36,Sc[getBlockIdx(36)]))) - (getDecValue(getBit(36,Sc[getBlockIdx(36)]))) - (getDecValue(getBit(37,Sc[getBlockIdx(37)]))) + getDecValue(getBit(38,Sc[getBlockIdx(38)])) - (getDecValue(getBit(43,Sc[getBlockIdx(43)]))) - (getDecValue(getBit(69,Sc[getBlockIdx(69)]))) + (short)2),&S[getBlockIdx(34)]);
    setBit(35,SIGN( + getDecValue(getBit(34,Sc[getBlockIdx(34)]))),&S[getBlockIdx(35)]);
    setBit(36,SIGN( - (getDecValue(getBit(4,Sc[getBlockIdx(4)]))) - (getDecValue(getBit(71,Sc[getBlockIdx(71)]))) + (short) 1),&S[getBlockIdx(36)]);
    setBit(37,SIGN( - (getDecValue(getBit(4,Sc[getBlockIdx(4)]))) + getDecValue(getBit(71,Sc[getBlockIdx(71)])) + (short) 1),&S[getBlockIdx(37)]);
    setBit(38,SIGN( - (getDecValue(getBit(8,Sc[getBlockIdx(8)]))) + getDecValue(getBit(39,Sc[getBlockIdx(39)])) + getDecValue(getBit(6,Sc[getBlockIdx(6)])) - (getDecValue(getBit(40,Sc[getBlockIdx(40)]))) - (getDecValue(getBit(63,Sc[getBlockIdx(63)]))) + (short)(-1)),&S[getBlockIdx(38)]);
    setBit(39,SIGN( + getDecValue(getBit(21,Sc[getBlockIdx(21)])) + getDecValue(getBit(23,Sc[getBlockIdx(23)])) - (getDecValue(getBit(34,Sc[getBlockIdx(34)]))) + getDecValue(getBit(49,Sc[getBlockIdx(49)])) + getDecValue(getBit(70,Sc[getBlockIdx(70)])) + getDecValue(getBit(84,Sc[getBlockIdx(84)])) + getDecValue(getBit(5,Sc[getBlockIdx(5)])) + (short)(-1)),&S[getBlockIdx(39)]);
    setBit(40,SIGN( + getDecValue(getBit(34,Sc[getBlockIdx(34)]))),&S[getBlockIdx(40)]);
    setBit(41,SIGN( - (getDecValue(getBit(22,Sc[getBlockIdx(22)]))) - (getDecValue(getBit(23,Sc[getBlockIdx(23)]))) - (getDecValue(getBit(24,Sc[getBlockIdx(24)]))) - (getDecValue(getBit(25,Sc[getBlockIdx(25)]))) + getDecValue(getBit(34,Sc[getBlockIdx(34)])) + getDecValue(getBit(43,Sc[getBlockIdx(43)])) + getDecValue(getBit(72,Sc[getBlockIdx(72)])) + (short) 1),&S[getBlockIdx(41)]);
    setBit(42,SIGN( - (getDecValue(getBit(41,Sc[getBlockIdx(41)]))) + (short) 1),&S[getBlockIdx(42)]);
    setBit(43,SIGN( + getDecValue(getBit(34,Sc[getBlockIdx(34)])) - (getDecValue(getBit(44,Sc[getBlockIdx(44)]))) - (getDecValue(getBit(60,Sc[getBlockIdx(60)]))) + getDecValue(getBit(88,Sc[getBlockIdx(88)])) + (short) 1),&S[getBlockIdx(43)]);
    setBit(44,SIGN( + getDecValue(getBit(21,Sc[getBlockIdx(21)])) + getDecValue(getBit(21,Sc[getBlockIdx(21)])) - (getDecValue(getBit(43,Sc[getBlockIdx(43)]))) - (getDecValue(getBit(45,Sc[getBlockIdx(45)]))) - (getDecValue(getBit(46,Sc[getBlockIdx(46)])))),&S[getBlockIdx(44)]);
    setBit(45,SIGN( - (getDecValue(getBit(34,Sc[getBlockIdx(34)]))) + getDecValue(getBit(43,Sc[getBlockIdx(43)])) - (getDecValue(getBit(44,Sc[getBlockIdx(44)]))) + getDecValue(getBit(68,Sc[getBlockIdx(68)]))),&S[getBlockIdx(45)]);
    setBit(46,SIGN( - (getDecValue(getBit(22,Sc[getBlockIdx(22)]))) - (getDecValue(getBit(24,Sc[getBlockIdx(24)]))) - (getDecValue(getBit(25,Sc[getBlockIdx(25)]))) - (getDecValue(getBit(34,Sc[getBlockIdx(34)]))) + (short) 1),&S[getBlockIdx(46)]);
    setBit(47,SIGN( - (getDecValue(getBit(43,Sc[getBlockIdx(43)]))) - (getDecValue(getBit(46,Sc[getBlockIdx(46)]))) + (short) 1),&S[getBlockIdx(47)]);
    setBit(48,SIGN( - (getDecValue(getBit(51,Sc[getBlockIdx(51)]))) - (getDecValue(getBit(52,Sc[getBlockIdx(52)]))) - (getDecValue(getBit(53,Sc[getBlockIdx(53)]))) - (getDecValue(getBit(54,Sc[getBlockIdx(54)]))) - (getDecValue(getBit(60,Sc[getBlockIdx(60)]))) + (short)2),&S[getBlockIdx(48)]);
    setBit(49,SIGN( - (getDecValue(getBit(48,Sc[getBlockIdx(48)]))) - (getDecValue(getBit(48,Sc[getBlockIdx(48)]))) - (getDecValue(getBit(51,Sc[getBlockIdx(51)]))) - (getDecValue(getBit(52,Sc[getBlockIdx(52)]))) + getDecValue(getBit(49,Sc[getBlockIdx(49)])) + (short) 1),&S[getBlockIdx(49)]);
    setBit(50,SIGN( + getDecValue(getBit(16,Sc[getBlockIdx(16)])) + getDecValue(getBit(38,Sc[getBlockIdx(38)])) + getDecValue(getBit(49,Sc[getBlockIdx(49)])) + (short)(-3)),&S[getBlockIdx(50)]);
    setBit(51,SIGN( + getDecValue(getBit(51,Sc[getBlockIdx(51)])) - (getDecValue(getBit(48,Sc[getBlockIdx(48)]))) - (getDecValue(getBit(56,Sc[getBlockIdx(56)]))) - (getDecValue(getBit(58,Sc[getBlockIdx(58)]))) - (getDecValue(getBit(59,Sc[getBlockIdx(59)]))) + getDecValue(getBit(80,Sc[getBlockIdx(80)])) + getDecValue(getBit(81,Sc[getBlockIdx(81)]))),&S[getBlockIdx(51)]);
    setBit(52,SIGN( - (getDecValue(getBit(43,Sc[getBlockIdx(43)]))) - (getDecValue(getBit(55,Sc[getBlockIdx(55)]))) - (getDecValue(getBit(56,Sc[getBlockIdx(56)]))) - (getDecValue(getBit(58,Sc[getBlockIdx(58)]))) - (getDecValue(getBit(59,Sc[getBlockIdx(59)]))) + (short) 1),&S[getBlockIdx(52)]);
    setBit(53,SIGN( + getDecValue(getBit(21,Sc[getBlockIdx(21)])) - (getDecValue(getBit(29,Sc[getBlockIdx(29)]))) - (getDecValue(getBit(29,Sc[getBlockIdx(29)]))) + getDecValue(getBit(38,Sc[getBlockIdx(38)])) - (getDecValue(getBit(58,Sc[getBlockIdx(58)]))) - (getDecValue(getBit(59,Sc[getBlockIdx(59)]))) - (getDecValue(getBit(64,Sc[getBlockIdx(64)]))) - (getDecValue(getBit(69,Sc[getBlockIdx(69)]))) + getDecValue(getBit(70,Sc[getBlockIdx(70)])) + getDecValue(getBit(84,Sc[getBlockIdx(84)])) + getDecValue(getBit(5,Sc[getBlockIdx(5)]))),&S[getBlockIdx(53)]);
    setBit(54,SIGN( - (getDecValue(getBit(48,Sc[getBlockIdx(48)]))) + getDecValue(getBit(49,Sc[getBlockIdx(49)])) - (getDecValue(getBit(51,Sc[getBlockIdx(51)]))) - (getDecValue(getBit(58,Sc[getBlockIdx(58)]))) - (getDecValue(getBit(59,Sc[getBlockIdx(59)])))),&S[getBlockIdx(54)]);
    setBit(55,SIGN( - (getDecValue(getBit(51,Sc[getBlockIdx(51)]))) - (getDecValue(getBit(52,Sc[getBlockIdx(52)]))) + getDecValue(getBit(56,Sc[getBlockIdx(56)])) + (short) 1),&S[getBlockIdx(55)]);
    setBit(56,SIGN( + getDecValue(getBit(52,Sc[getBlockIdx(52)])) - (getDecValue(getBit(55,Sc[getBlockIdx(55)])))),&S[getBlockIdx(56)]);
    setBit(57,SIGN( + getDecValue(getBit(51,Sc[getBlockIdx(51)])) + getDecValue(getBit(52,Sc[getBlockIdx(52)])) - (getDecValue(getBit(55,Sc[getBlockIdx(55)]))) + getDecValue(getBit(56,Sc[getBlockIdx(56)])) + getDecValue(getBit(57,Sc[getBlockIdx(57)]))),&S[getBlockIdx(57)]);
    setBit(58,SIGN( - (getDecValue(getBit(25,Sc[getBlockIdx(25)]))) + getDecValue(getBit(34,Sc[getBlockIdx(34)])) - (getDecValue(getBit(38,Sc[getBlockIdx(38)]))) - (getDecValue(getBit(51,Sc[getBlockIdx(51)]))) - (getDecValue(getBit(52,Sc[getBlockIdx(52)]))) - (getDecValue(getBit(53,Sc[getBlockIdx(53)]))) + getDecValue(getBit(62,Sc[getBlockIdx(62)])) + (short) 1),&S[getBlockIdx(58)]);
    setBit(59,SIGN( - (getDecValue(getBit(25,Sc[getBlockIdx(25)]))) + getDecValue(getBit(34,Sc[getBlockIdx(34)])) - (getDecValue(getBit(38,Sc[getBlockIdx(38)]))) + getDecValue(getBit(43,Sc[getBlockIdx(43)])) + getDecValue(getBit(62,Sc[getBlockIdx(62)])) - (getDecValue(getBit(78,Sc[getBlockIdx(78)]))) + (short) 1),&S[getBlockIdx(59)]);
    setBit(60,SIGN( + getDecValue(getBit(25,Sc[getBlockIdx(25)])) + getDecValue(getBit(43,Sc[getBlockIdx(43)])) - (getDecValue(getBit(50,Sc[getBlockIdx(50)]))) - (getDecValue(getBit(87,Sc[getBlockIdx(87)]))) + (short) 1),&S[getBlockIdx(60)]);
    setBit(61,SIGN( + getDecValue(getBit(3,Sc[getBlockIdx(3)])) + getDecValue(getBit(8,Sc[getBlockIdx(8)]))),&S[getBlockIdx(61)]);
    setBit(62,SIGN( + getDecValue(getBit(61,Sc[getBlockIdx(61)])) + getDecValue(getBit(85,Sc[getBlockIdx(85)])) + (short)(-1)),&S[getBlockIdx(62)]);
    setBit(63,SIGN( + getDecValue(getBit(61,Sc[getBlockIdx(61)]))),&S[getBlockIdx(63)]);
    setBit(64,SIGN( + getDecValue(getBit(62,Sc[getBlockIdx(62)])) + getDecValue(getBit(85,Sc[getBlockIdx(85)]))),&S[getBlockIdx(64)]);
    setBit(65,SIGN( + getDecValue(getBit(3,Sc[getBlockIdx(3)]))),&S[getBlockIdx(65)]);
    setBit(66,SIGN( + getDecValue(getBit(65,Sc[getBlockIdx(65)]))),&S[getBlockIdx(66)]);
    setBit(67,SIGN( + getDecValue(getBit(66,Sc[getBlockIdx(66)]))),&S[getBlockIdx(67)]);
    setBit(68,SIGN( + getDecValue(getBit(8,Sc[getBlockIdx(8)]))),&S[getBlockIdx(68)]);
    setBit(69,SIGN( - (getDecValue(getBit(25,Sc[getBlockIdx(25)]))) + (short)2),&S[getBlockIdx(69)]);
    setBit(70,SIGN( + getDecValue(getBit(23,Sc[getBlockIdx(23)])) + getDecValue(getBit(68,Sc[getBlockIdx(68)]))),&S[getBlockIdx(70)]);
    setBit(71,SIGN( - (getDecValue(getBit(35,Sc[getBlockIdx(35)]))) - (getDecValue(getBit(83,Sc[getBlockIdx(83)])))),&S[getBlockIdx(71)]);
    setBit(72,SIGN( - (getDecValue(getBit(1,Sc[getBlockIdx(1)]))) + getDecValue(getBit(11,Sc[getBlockIdx(11)])) + getDecValue(getBit(34,Sc[getBlockIdx(34)])) + getDecValue(getBit(87,Sc[getBlockIdx(87)])) + (short) 1),&S[getBlockIdx(72)]);
    setBit(73,SIGN( - (getDecValue(getBit(25,Sc[getBlockIdx(25)]))) + getDecValue(getBit(43,Sc[getBlockIdx(43)])) - (getDecValue(getBit(44,Sc[getBlockIdx(44)]))) + getDecValue(getBit(45,Sc[getBlockIdx(45)])) - (getDecValue(getBit(47,Sc[getBlockIdx(47)]))) + getDecValue(getBit(66,Sc[getBlockIdx(66)])) + getDecValue(getBit(67,Sc[getBlockIdx(67)]))),&S[getBlockIdx(73)]);
    setBit(74,SIGN( + getDecValue(getBit(73,Sc[getBlockIdx(73)]))),&S[getBlockIdx(74)]);
    setBit(75,SIGN( + getDecValue(getBit(66,Sc[getBlockIdx(66)])) + getDecValue(getBit(74,Sc[getBlockIdx(74)]))),&S[getBlockIdx(75)]);
    setBit(76,SIGN( - (getDecValue(getBit(21,Sc[getBlockIdx(21)]))) - (getDecValue(getBit(32,Sc[getBlockIdx(32)]))) - (getDecValue(getBit(95,Sc[getBlockIdx(95)]))) + (short)3),&S[getBlockIdx(76)]);
    setBit(77,SIGN( + getDecValue(getBit(25,Sc[getBlockIdx(25)])) + getDecValue(getBit(34,Sc[getBlockIdx(34)])) + getDecValue(getBit(38,Sc[getBlockIdx(38)])) + (short)(-1)),&S[getBlockIdx(77)]);
    setBit(78,SIGN( + getDecValue(getBit(12,Sc[getBlockIdx(12)])) + getDecValue(getBit(21,Sc[getBlockIdx(21)])) + getDecValue(getBit(25,Sc[getBlockIdx(25)])) + getDecValue(getBit(34,Sc[getBlockIdx(34)])) + getDecValue(getBit(38,Sc[getBlockIdx(38)])) - (getDecValue(getBit(43,Sc[getBlockIdx(43)]))) - (getDecValue(getBit(62,Sc[getBlockIdx(62)]))) - (getDecValue(getBit(91,Sc[getBlockIdx(91)]))) + (short)(-4)),&S[getBlockIdx(78)]);
    setBit(79,SIGN( + getDecValue(getBit(34,Sc[getBlockIdx(34)])) + getDecValue(getBit(38,Sc[getBlockIdx(38)]))),&S[getBlockIdx(79)]);
    setBit(80,SIGN( + getDecValue(getBit(49,Sc[getBlockIdx(49)])) + getDecValue(getBit(54,Sc[getBlockIdx(54)])) + (short)(-1)),&S[getBlockIdx(80)]);
    setBit(81,SIGN( + getDecValue(getBit(55,Sc[getBlockIdx(55)])) + getDecValue(getBit(57,Sc[getBlockIdx(57)])) + (short)(-1)),&S[getBlockIdx(81)]);
    setBit(82,SIGN( + getDecValue(getBit(3,Sc[getBlockIdx(3)]))),&S[getBlockIdx(82)]);
    setBit(83,SIGN( + getDecValue(getBit(21,Sc[getBlockIdx(21)])) + getDecValue(getBit(38,Sc[getBlockIdx(38)])) + getDecValue(getBit(59,Sc[getBlockIdx(59)]))),&S[getBlockIdx(83)]);
    setBit(84,SIGN( + getDecValue(getBit(31,Sc[getBlockIdx(31)])) - (getDecValue(getBit(82,Sc[getBlockIdx(82)])))),&S[getBlockIdx(84)]);
    setBit(85,SIGN( - (getDecValue(getBit(38,Sc[getBlockIdx(38)]))) + (short) 1),&S[getBlockIdx(85)]);
    setBit(86,SIGN( + getDecValue(getBit(19,Sc[getBlockIdx(19)])) + getDecValue(getBit(33,Sc[getBlockIdx(33)]))),&S[getBlockIdx(86)]);
    setBit(87,SIGN( + getDecValue(getBit(9,Sc[getBlockIdx(9)]))),&S[getBlockIdx(87)]);
    setBit(88,SIGN( + getDecValue(getBit(87,Sc[getBlockIdx(87)]))),&S[getBlockIdx(88)]);
    setBit(89,SIGN( + getDecValue(getBit(87,Sc[getBlockIdx(87)]))),&S[getBlockIdx(89)]);
    setBit(90,SIGN( + getDecValue(getBit(24,Sc[getBlockIdx(24)])) + getDecValue(getBit(86,Sc[getBlockIdx(86)]))),&S[getBlockIdx(90)]);
    setBit(91,SIGN( - (getDecValue(getBit(90,Sc[getBlockIdx(90)]))) + (short) 1),&S[getBlockIdx(91)]);
    setBit(92,SIGN( + getDecValue(getBit(7,Sc[getBlockIdx(7)])) + getDecValue(getBit(43,Sc[getBlockIdx(43)])) + (short)(-1)),&S[getBlockIdx(92)]);
    setBit(93,SIGN( + getDecValue(getBit(34,Sc[getBlockIdx(34)])) + getDecValue(getBit(38,Sc[getBlockIdx(38)])) + (short)(-1)),&S[getBlockIdx(93)]);
    setBit(94,SIGN( + getDecValue(getBit(93,Sc[getBlockIdx(93)]))),&S[getBlockIdx(94)]);
    setBit(95,SIGN( + getDecValue(getBit(21,Sc[getBlockIdx(21)])) - (getDecValue(getBit(29,Sc[getBlockIdx(29)]))) - (getDecValue(getBit(43,Sc[getBlockIdx(43)]))) + getDecValue(getBit(61,Sc[getBlockIdx(61)])) + (short)(-1)),&S[getBlockIdx(95)]);
}


__global__
void findAttractor(uint32 *attractors, uint32 *transients, uint32 *periods, uint32_t numThreads){
    int transient = 0, period = 0;
    uint32 S0[N], S1[N];
    uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t step =  NUM_STATES / NUM_COPYS; 
    uint32_t rest =  NUM_STATES % NUM_COPYS;
    uint32_t begin = 0;
    uint32_t end = step - 1;
    
    hiprandState_t state;
	hiprand_init(thread, /* the seed controls the sequence of random values that are produced */
              0, /* the sequence number is only important with multiple cores */
              1, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &state);
              
    bool flag = true;
    if(thread < numThreads){
        if(rest > 0){
            end = end + 1;
            rest = rest - 1;
        }else{
        	flag = false;
        }
        
        
        for(uint32_t i = 0; i < NUM_COPYS;i++){
            if(i == thread) break;
            
			if(rest > 0){
                end = end + 1;
                begin = begin + 1;
                rest = rest - 1;
            }else if(rest == 0 && flag){
                begin = begin + 1;
                flag = 0;
            }
            begin += step;
            end += step;
        }

	    for (uint32 i = begin; i <= end; ++i) {
	        S0[0] = S1[0] = hiprand(&state);
            S0[1] = S1[1] = hiprand(&state);
            S0[2] = S1[2] = hiprand(&state);
            setBit(0, 1, &S0[getBlockIdx(0)]);      // garante as entradas fixas
            setBit(1, 1, &S0[getBlockIdx(1)]);
            setBit(2, 0, &S0[getBlockIdx(2)]);
            setBit(3, 0, &S0[getBlockIdx(3)]);
            setBit(4, 0, &S0[getBlockIdx(4)]);
            setBit(5, 0, &S0[getBlockIdx(5)]);
            setBit(6, 1, &S0[getBlockIdx(6)]);
            setBit(7, 1, &S0[getBlockIdx(7)]);
            setBit(0, 1, &S1[getBlockIdx(0)]);
            setBit(1, 1, &S1[getBlockIdx(1)]);
            setBit(2, 0, &S1[getBlockIdx(2)]);
            setBit(3, 0, &S1[getBlockIdx(3)]);
            setBit(4, 0, &S1[getBlockIdx(4)]);
            setBit(5, 0, &S1[getBlockIdx(5)]);
            setBit(6, 1, &S1[getBlockIdx(6)]);
            setBit(7, 1, &S1[getBlockIdx(7)]);
            
            transient = 0;
            period = 0;
            do{
                pass(S0);
                pass(S0);
                pass(S1);
                transient++;
            }while(!comp(S0,S1));
            do{
                pass(S0);
                period++;
            }while (!comp(S0,S1));
            period--;
            //printf("%u %u, %u , %u, Trans %u, Per %u\n",i,S0[0], S0[1], S0[2],transient,period);
	        
            transients[i] = transient;
	        periods[i]= period;
	        for(int s = 0; s < N; s++){
	            attractors[(i * N) + s] = S0[s];
	        }
	        //cout << transient << " " << period << std::endl;
	    }
	}	
}

int main() {
	// Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    // Print the vector length to be used, and compute its size
    size_t numState = NUM_STATES;
    size_t size = N * numState * sizeof(uint32);
    size_t size_transients = numState*sizeof(uint32);
    size_t size_periods = numState*sizeof(uint32);
    size_t totalBytes = size+size_transients+size_periods;
    size_t kb = totalBytes/(1024);
    size_t mb = kb/(1024);
    size_t gb = mb/(1024);
    printf("Find attractors net %d nodes in %lu initials states.\n", N,numState);
    printf("Memory usage: %lu Gb or %lu Mb or %lu Kb.\n", gb, mb, kb);
    
	uint32 *h_transients = (uint32*)malloc(size_transients);
    // Verifica se houve sucesso na aloca��o do vetor h_transients
    if (h_transients == NULL){
        fprintf(stderr, "Failed to allocate h_transients!\n");
        exit(EXIT_FAILURE);
    }    
    
    uint32 *h_periods = (uint32*)malloc(size_periods);
    // Verifica se houve sucesso na aloca��o do vetor h_periods
    if (h_periods == NULL){
        fprintf(stderr, "Failed to allocate h_periods!\n");
        exit(EXIT_FAILURE);
    }
    
    //Aloca o vetor para a saida no host
    uint32 *h_attractors = (uint32*)malloc(size);
    // Verifica se houve sucesso na aloca��o do vetor h_attractors
    if (h_attractors == NULL){
        fprintf(stderr, "Failed to allocate h_attractors!\n");
        exit(EXIT_FAILURE);
    }
    
    // Aloca os vetores na GPU (device)
    uint32 *d_transients = NULL;
    err = hipMalloc((void **)&d_transients, NUM_STATES * sizeof(uint32));

    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate d_transients (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    uint32 *d_periods = NULL;
    err = hipMalloc((void **)&d_periods, NUM_STATES * sizeof(uint32));

    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate d_periods (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    uint32 *d_attractors = NULL;
    err = hipMalloc((void **)&d_attractors, size);

    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate d_attractors (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    uint32_t threadsPerBlock = 256;
    uint32_t blocksPerGrid = (NUM_COPYS + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    
    findAttractor<<< blocksPerGrid, threadsPerBlock >>>(d_attractors, d_transients, d_periods, NUM_COPYS);
    //findAttractor<<< 1,1 >>>(d_attractors, d_transients, d_periods, NUM_COPYS);
    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch findAttractor kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    
	err = hipMemcpy(h_transients, d_transients, size_transients, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector d_transients from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(h_periods, d_periods, size_periods, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector d_periods from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(h_attractors, d_attractors, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector d_attractors from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    ///*
//    printf("Attractor found:\n");
//      
//    
//     for(int i = 0; i < numState; i++){
//        for(int j = 0; j < N; j++){
//           printf("%u   ", h_attractors[(i * N) + j]);
//       }
//       printf(" Trans:%u Per:%u\n", h_transients[i],h_periods[i]);
//     }
//     printf("\n");
    //*/
    err = hipFree(d_transients);
    
    if (err != hipSuccess){
        fprintf(stderr, "Failed to free device vector d_transients (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipFree(d_periods);

    if (err != hipSuccess){
        fprintf(stderr, "Failed to free device vector d_periods (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipFree(d_attractors);

    if (err != hipSuccess){
        fprintf(stderr, "Failed to free device vector d_attractors (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_transients);
    free(h_periods);
    free(h_attractors);

    // Reset the device and exit
    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess){
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}
