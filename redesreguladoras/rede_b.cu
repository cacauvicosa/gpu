#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define SIGN(x) (x > 1) ? 1 : ((x < -1) ? -1 : x)
#define NUM_STATES (1<<20)
#define NUM_COPYS  (1 << 10)
#define NUM_NOS 96
#define N 3

using namespace std;
typedef unsigned long uint64;

__device__
bool comp(uint64 *S0, uint64 *S1){
    for (int i = 0; i < N; ++i) {
        if(S1[i] != S0[i])
            return false;
    }
    return true;
}

__device__
short getDecValue(short v){
    short vp = 0;
    switch(v){
        case 0:
        case 2: vp = 0;
            break;
        case 1: vp = 1;
            break;
        case 3: vp = -1;
            break;
    }
    return vp;
}

__device__
short get2bit(short idx, uint64 v){
    idx = idx*2;
    return (v >> idx) & 3;
}

__device__
void set2bit(short idx, short newV, uint64 *v){
    uint64 mask = 3;
    idx = idx*2;
    newV = (newV == -1) ? 3 : newV;
    *v &= ~(mask << idx);
    *v |= ((uint64) newV << idx);
}

__device__
short getBlockIdx(short idx){
    idx = idx*2;
    if(idx<64)
        return 0;
    else if(idx>127)
        return 2;
    else
        return 1;
}

__device__
void pass(uint64 *S) {
    uint64 Sc[N];
    for (int i = 0; i < N; ++i) {
        Sc[i] = S[i];
    }
    set2bit(0,1, &S[getBlockIdx(0)]);
    set2bit(1,1, &S[getBlockIdx(1)]);
    set2bit(2,-1, &S[getBlockIdx(2)]);
    set2bit(3,0, &S[getBlockIdx(3)]);
    set2bit(4,-1, &S[getBlockIdx(4)]);
    set2bit(5,0, &S[getBlockIdx(5)]);
    set2bit(6,1, &S[getBlockIdx(6)]);
    set2bit(7,1, &S[getBlockIdx(7)]);
    set2bit(8,SIGN( + getDecValue(get2bit(34, Sc[getBlockIdx(34)]))), &S[getBlockIdx(8)]);
    set2bit(9,SIGN( + getDecValue(get2bit(0, Sc[getBlockIdx(0)])) + getDecValue(get2bit(71, Sc[getBlockIdx(71)]))), &S[getBlockIdx(9)]);
    set2bit(10,SIGN( + getDecValue(get2bit(43, Sc[getBlockIdx(43)])) + getDecValue(get2bit(60, Sc[getBlockIdx(60)])) + (-1)), &S[getBlockIdx(10)]);
    set2bit(11,SIGN( - (getDecValue(get2bit(2, Sc[getBlockIdx(2)]))) + 1), &S[getBlockIdx(11)]);
    set2bit(12,SIGN( - (getDecValue(get2bit(13, Sc[getBlockIdx(13)]))) + 1), &S[getBlockIdx(12)]);
    set2bit(13,SIGN( + getDecValue(get2bit(14, Sc[getBlockIdx(14)])) + getDecValue(get2bit(26, Sc[getBlockIdx(26)]))), &S[getBlockIdx(13)]);
    set2bit(14,SIGN( + getDecValue(get2bit(1, Sc[getBlockIdx(1)]))), &S[getBlockIdx(14)]);
    set2bit(15,SIGN( + getDecValue(get2bit(2, Sc[getBlockIdx(2)])) - (getDecValue(get2bit(4,Sc[getBlockIdx(4)])))),&S[getBlockIdx(15)]);
    set2bit(16,SIGN( - (getDecValue(get2bit(12, Sc[getBlockIdx(12)]))) + getDecValue(get2bit(14,Sc[getBlockIdx(14)])) + 1),&S[getBlockIdx(16)]);
    set2bit(17,SIGN( + getDecValue(get2bit(16, Sc[getBlockIdx(16)])) + getDecValue(get2bit(78,Sc[getBlockIdx(78)]))),&S[getBlockIdx(17)]);
    set2bit(18,SIGN( + getDecValue(get2bit(17, Sc[getBlockIdx(17)])) - (getDecValue(get2bit(7,Sc[getBlockIdx(7)]))) - (getDecValue(get2bit(92, Sc[getBlockIdx(92)]))) + 1), &S[getBlockIdx(18)]);
    set2bit(19,SIGN( + getDecValue(get2bit(18, Sc[getBlockIdx(18)])) + getDecValue(get2bit(34,Sc[getBlockIdx(34)])) + getDecValue(get2bit(38, Sc[getBlockIdx(38)]))), &S[getBlockIdx(19)]);
    set2bit(20,SIGN( + getDecValue(get2bit(13,Sc[getBlockIdx(13)])) + getDecValue(get2bit(25,Sc[getBlockIdx(25)])) + getDecValue(get2bit(33, Sc[getBlockIdx(33)])) - (getDecValue(get2bit(37,Sc[getBlockIdx(37)]))) - (getDecValue(get2bit(43,Sc[getBlockIdx(43)]))) + getDecValue(get2bit(82,Sc[getBlockIdx(82)]))),&S[getBlockIdx(20)]);
    set2bit(21,SIGN( + getDecValue(get2bit(18,Sc[getBlockIdx(18)])) + getDecValue(get2bit(20,Sc[getBlockIdx(20)])) + getDecValue(get2bit(20, Sc[getBlockIdx(20)])) - (getDecValue(get2bit(76,Sc[getBlockIdx(76)]))) + getDecValue(get2bit(95,Sc[getBlockIdx(95)])) + (-1)),&S[getBlockIdx(21)]);
    set2bit(22,SIGN( + getDecValue(get2bit(13,Sc[getBlockIdx(13)])) + getDecValue(get2bit(16,Sc[getBlockIdx(16)]))), &S[getBlockIdx(22)]);
    set2bit(23,SIGN( + getDecValue(get2bit(22,Sc[getBlockIdx(22)]))), &S[getBlockIdx(23)]);
    set2bit(24,SIGN( + getDecValue(get2bit(19,Sc[getBlockIdx(19)])) + getDecValue(get2bit(23,Sc[getBlockIdx(23)]))),&S[getBlockIdx(24)]);
    set2bit(25,SIGN( + getDecValue(get2bit(18,Sc[getBlockIdx(18)])) + getDecValue(get2bit(19,Sc[getBlockIdx(19)])) + (-1)),&S[getBlockIdx(25)]);
    set2bit(26,SIGN( - (getDecValue(get2bit(43,Sc[getBlockIdx(43)]))) + getDecValue(get2bit(5,Sc[getBlockIdx(5)]))),&S[getBlockIdx(26)]);
    set2bit(27,SIGN( + getDecValue(get2bit(26,Sc[getBlockIdx(26)]))), &S[getBlockIdx(27)]);
    set2bit(28,SIGN( + getDecValue(get2bit(7,Sc[getBlockIdx(7)])) + 1), &S[getBlockIdx(28)]);
    set2bit(29,SIGN( - (getDecValue(get2bit(24,Sc[getBlockIdx(24)]))) - (getDecValue(get2bit(25,Sc[getBlockIdx(25)]))) - (getDecValue(get2bit(27,Sc[getBlockIdx(27)]))) - (getDecValue(get2bit(33,Sc[getBlockIdx(33)]))) + 3),&S[getBlockIdx(29)]);
    set2bit(30,SIGN( + getDecValue(get2bit(28,Sc[getBlockIdx(28)])) + getDecValue(get2bit(29,Sc[getBlockIdx(29)])) + (-1)),&S[getBlockIdx(30)]);
    set2bit(31,SIGN( - (getDecValue(get2bit(30,Sc[getBlockIdx(30)]))) - (getDecValue(get2bit(43,Sc[getBlockIdx(43)]))) + 1),&S[getBlockIdx(31)]);
    set2bit(32,SIGN( - (getDecValue(get2bit(10,Sc[getBlockIdx(10)]))) + getDecValue(get2bit(21,Sc[getBlockIdx(21)])) + getDecValue(get2bit(84,Sc[getBlockIdx(84)]))),&S[getBlockIdx(32)]);
    set2bit(33,SIGN( + getDecValue(get2bit(15,Sc[getBlockIdx(15)])) + getDecValue(get2bit(25,Sc[getBlockIdx(25)])) + getDecValue(get2bit(42,Sc[getBlockIdx(42)])) - (getDecValue(get2bit(72,Sc[getBlockIdx(72)]))) + (-1)),&S[getBlockIdx(33)]);
    set2bit(34,SIGN( + getDecValue(get2bit(4,Sc[getBlockIdx(4)])) + getDecValue(get2bit(33,Sc[getBlockIdx(33)])) - (getDecValue(get2bit(36,Sc[getBlockIdx(36)]))) - (getDecValue(get2bit(36,Sc[getBlockIdx(36)]))) - (getDecValue(get2bit(37,Sc[getBlockIdx(37)]))) + getDecValue(get2bit(38,Sc[getBlockIdx(38)])) - (getDecValue(get2bit(43,Sc[getBlockIdx(43)]))) - (getDecValue(get2bit(69,Sc[getBlockIdx(69)]))) + 2),&S[getBlockIdx(34)]);
    set2bit(35,SIGN( + getDecValue(get2bit(34,Sc[getBlockIdx(34)]))), &S[getBlockIdx(35)]);
    set2bit(36,SIGN( - (getDecValue(get2bit(4,Sc[getBlockIdx(4)]))) - (getDecValue(get2bit(71,Sc[getBlockIdx(71)]))) + 1),&S[getBlockIdx(36)]);
    set2bit(37,SIGN( - (getDecValue(get2bit(4,Sc[getBlockIdx(4)]))) + getDecValue(get2bit(71,Sc[getBlockIdx(71)])) + 1),&S[getBlockIdx(37)]);
    set2bit(38,SIGN( - (getDecValue(get2bit(8,Sc[getBlockIdx(8)]))) + getDecValue(get2bit(39,Sc[getBlockIdx(39)])) + getDecValue(get2bit(6,Sc[getBlockIdx(6)])) - (getDecValue(get2bit(40,Sc[getBlockIdx(40)]))) - (getDecValue(get2bit(63,Sc[getBlockIdx(63)]))) + (-1)),&S[getBlockIdx(38)]);
    set2bit(39,SIGN( + getDecValue(get2bit(21,Sc[getBlockIdx(21)])) + getDecValue(get2bit(23,Sc[getBlockIdx(23)])) - (getDecValue(get2bit(34,Sc[getBlockIdx(34)]))) + getDecValue(get2bit(49,Sc[getBlockIdx(49)])) + getDecValue(get2bit(70,Sc[getBlockIdx(70)])) + getDecValue(get2bit(84,Sc[getBlockIdx(84)])) + getDecValue(get2bit(5,Sc[getBlockIdx(5)])) + (-1)),&S[getBlockIdx(39)]);
    set2bit(40,SIGN( + getDecValue(get2bit(34,Sc[getBlockIdx(34)]))),&S[getBlockIdx(40)]);
    set2bit(41,SIGN( - (getDecValue(get2bit(22,Sc[getBlockIdx(22)]))) - (getDecValue(get2bit(23,Sc[getBlockIdx(23)]))) - (getDecValue(get2bit(24,Sc[getBlockIdx(24)]))) - (getDecValue(get2bit(25,Sc[getBlockIdx(25)]))) + getDecValue(get2bit(34,Sc[getBlockIdx(34)])) + getDecValue(get2bit(43,Sc[getBlockIdx(43)])) + getDecValue(get2bit(72,Sc[getBlockIdx(72)])) + 1),&S[getBlockIdx(41)]);
    set2bit(42,SIGN( - (getDecValue(get2bit(41,Sc[getBlockIdx(41)]))) + 1), &S[getBlockIdx(42)]);
    set2bit(43,SIGN( + getDecValue(get2bit(34,Sc[getBlockIdx(34)])) - (getDecValue(get2bit(44,Sc[getBlockIdx(44)]))) - (getDecValue(get2bit(60,Sc[getBlockIdx(60)]))) + getDecValue(get2bit(88,Sc[getBlockIdx(88)])) + 1),&S[getBlockIdx(43)]);
    set2bit(44,SIGN( + getDecValue(get2bit(21,Sc[getBlockIdx(21)])) + getDecValue(get2bit(21,Sc[getBlockIdx(21)])) - (getDecValue(get2bit(43,Sc[getBlockIdx(43)]))) - (getDecValue(get2bit(45,Sc[getBlockIdx(45)]))) - (getDecValue(get2bit(46,Sc[getBlockIdx(46)])))),&S[getBlockIdx(44)]);
    set2bit(45,SIGN( - (getDecValue(get2bit(34,Sc[getBlockIdx(34)]))) + getDecValue(get2bit(43,Sc[getBlockIdx(43)])) - (getDecValue(get2bit(44,Sc[getBlockIdx(44)]))) + getDecValue(get2bit(68,Sc[getBlockIdx(68)]))),&S[getBlockIdx(45)]);
    set2bit(46,SIGN( - (getDecValue(get2bit(22,Sc[getBlockIdx(22)]))) - (getDecValue(get2bit(24,Sc[getBlockIdx(24)]))) - (getDecValue(get2bit(25,Sc[getBlockIdx(25)]))) - (getDecValue(get2bit(34,Sc[getBlockIdx(34)]))) + 1),&S[getBlockIdx(46)]);
    set2bit(47,SIGN( - (getDecValue(get2bit(43,Sc[getBlockIdx(43)]))) - (getDecValue(get2bit(46,Sc[getBlockIdx(46)]))) + 1),&S[getBlockIdx(47)]);
    set2bit(48,SIGN( - (getDecValue(get2bit(51,Sc[getBlockIdx(51)]))) - (getDecValue(get2bit(52,Sc[getBlockIdx(52)]))) - (getDecValue(get2bit(53,Sc[getBlockIdx(53)]))) - (getDecValue(get2bit(54,Sc[getBlockIdx(54)]))) - (getDecValue(get2bit(60,Sc[getBlockIdx(60)]))) + 2),&S[getBlockIdx(48)]);
    set2bit(49,SIGN( - (getDecValue(get2bit(48,Sc[getBlockIdx(48)]))) - (getDecValue(get2bit(48,Sc[getBlockIdx(48)]))) - (getDecValue(get2bit(51,Sc[getBlockIdx(51)]))) - (getDecValue(get2bit(52,Sc[getBlockIdx(52)]))) + getDecValue(get2bit(49,Sc[getBlockIdx(49)])) + 1),&S[getBlockIdx(49)]);
    set2bit(50,SIGN( + getDecValue(get2bit(16,Sc[getBlockIdx(16)])) + getDecValue(get2bit(38,Sc[getBlockIdx(38)])) + getDecValue(get2bit(49,Sc[getBlockIdx(49)])) + (-3)),&S[getBlockIdx(50)]);
    set2bit(51,SIGN( + getDecValue(get2bit(51,Sc[getBlockIdx(51)])) - (getDecValue(get2bit(48,Sc[getBlockIdx(48)]))) - (getDecValue(get2bit(56,Sc[getBlockIdx(56)]))) - (getDecValue(get2bit(58,Sc[getBlockIdx(58)]))) - (getDecValue(get2bit(59,Sc[getBlockIdx(59)]))) + getDecValue(get2bit(80,Sc[getBlockIdx(80)])) + getDecValue(get2bit(81,Sc[getBlockIdx(81)]))),&S[getBlockIdx(51)]);
    set2bit(52,SIGN( - (getDecValue(get2bit(43,Sc[getBlockIdx(43)]))) - (getDecValue(get2bit(55,Sc[getBlockIdx(55)]))) - (getDecValue(get2bit(56,Sc[getBlockIdx(56)]))) - (getDecValue(get2bit(58,Sc[getBlockIdx(58)]))) - (getDecValue(get2bit(59,Sc[getBlockIdx(59)]))) + 1),&S[getBlockIdx(52)]);
    set2bit(53,SIGN( + getDecValue(get2bit(21,Sc[getBlockIdx(21)])) - (getDecValue(get2bit(29,Sc[getBlockIdx(29)]))) - (getDecValue(get2bit(29,Sc[getBlockIdx(29)]))) + getDecValue(get2bit(38,Sc[getBlockIdx(38)])) - (getDecValue(get2bit(58,Sc[getBlockIdx(58)]))) - (getDecValue(get2bit(59,Sc[getBlockIdx(59)]))) - (getDecValue(get2bit(64,Sc[getBlockIdx(64)]))) - (getDecValue(get2bit(69,Sc[getBlockIdx(69)]))) + getDecValue(get2bit(70,Sc[getBlockIdx(70)])) + getDecValue(get2bit(84,Sc[getBlockIdx(84)])) + getDecValue(get2bit(5,Sc[getBlockIdx(5)]))),&S[getBlockIdx(53)]);
    set2bit(54,SIGN( - (getDecValue(get2bit(48,Sc[getBlockIdx(48)]))) + getDecValue(get2bit(49,Sc[getBlockIdx(49)])) - (getDecValue(get2bit(51,Sc[getBlockIdx(51)]))) - (getDecValue(get2bit(58,Sc[getBlockIdx(58)]))) - (getDecValue(get2bit(59,Sc[getBlockIdx(59)])))),&S[getBlockIdx(54)]);
    set2bit(55,SIGN( - (getDecValue(get2bit(51,Sc[getBlockIdx(51)]))) - (getDecValue(get2bit(52,Sc[getBlockIdx(52)]))) + getDecValue(get2bit(56,Sc[getBlockIdx(56)])) + 1),&S[getBlockIdx(55)]);
    set2bit(56,SIGN( + getDecValue(get2bit(52,Sc[getBlockIdx(52)])) - (getDecValue(get2bit(55,Sc[getBlockIdx(55)])))), &S[getBlockIdx(56)]);
    set2bit(57,SIGN( + getDecValue(get2bit(51,Sc[getBlockIdx(51)])) + getDecValue(get2bit(52,Sc[getBlockIdx(52)])) - (getDecValue(get2bit(55,Sc[getBlockIdx(55)]))) + getDecValue(get2bit(56,Sc[getBlockIdx(56)])) + getDecValue(get2bit(57,Sc[getBlockIdx(57)]))),&S[getBlockIdx(57)]);
    set2bit(58,SIGN( - (getDecValue(get2bit(25,Sc[getBlockIdx(25)]))) + getDecValue(get2bit(34,Sc[getBlockIdx(34)])) - (getDecValue(get2bit(38,Sc[getBlockIdx(38)]))) - (getDecValue(get2bit(51,Sc[getBlockIdx(51)]))) - (getDecValue(get2bit(52,Sc[getBlockIdx(52)]))) - (getDecValue(get2bit(53,Sc[getBlockIdx(53)]))) + getDecValue(get2bit(62,Sc[getBlockIdx(62)])) + 1),&S[getBlockIdx(58)]);
    set2bit(59,SIGN( - (getDecValue(get2bit(25,Sc[getBlockIdx(25)]))) + getDecValue(get2bit(34,Sc[getBlockIdx(34)])) - (getDecValue(get2bit(38,Sc[getBlockIdx(38)]))) + getDecValue(get2bit(43,Sc[getBlockIdx(43)])) + getDecValue(get2bit(62,Sc[getBlockIdx(62)])) - (getDecValue(get2bit(78,Sc[getBlockIdx(78)]))) + 1),&S[getBlockIdx(59)]);
    set2bit(60,SIGN( + getDecValue(get2bit(25,Sc[getBlockIdx(25)])) + getDecValue(get2bit(43,Sc[getBlockIdx(43)])) - (getDecValue(get2bit(50,Sc[getBlockIdx(50)]))) - (getDecValue(get2bit(87,Sc[getBlockIdx(87)]))) + 1),&S[getBlockIdx(60)]);
    set2bit(61,SIGN( + getDecValue(get2bit(3,Sc[getBlockIdx(3)])) + getDecValue(get2bit(8,Sc[getBlockIdx(8)]))),&S[getBlockIdx(61)]);
    set2bit(62,SIGN( + getDecValue(get2bit(61,Sc[getBlockIdx(61)])) + getDecValue(get2bit(85,Sc[getBlockIdx(85)])) + (-1)),&S[getBlockIdx(62)]);
    set2bit(63,SIGN( + getDecValue(get2bit(61,Sc[getBlockIdx(61)]))), &S[getBlockIdx(63)]);
    set2bit(64,SIGN( + getDecValue(get2bit(62,Sc[getBlockIdx(62)])) + getDecValue(get2bit(85,Sc[getBlockIdx(85)]))),&S[getBlockIdx(64)]);
    set2bit(65,SIGN( + getDecValue(get2bit(3,Sc[getBlockIdx(3)]))), &S[getBlockIdx(65)]);
    set2bit(66,SIGN( + getDecValue(get2bit(65,Sc[getBlockIdx(65)]))), &S[getBlockIdx(66)]);
    set2bit(67,SIGN( + getDecValue(get2bit(66,Sc[getBlockIdx(66)]))), &S[getBlockIdx(67)]);
    set2bit(68,SIGN( + getDecValue(get2bit(8,Sc[getBlockIdx(8)]))),&S[getBlockIdx(68)]);
    set2bit(69,SIGN( - (getDecValue(get2bit(25,Sc[getBlockIdx(25)]))) + 2), &S[getBlockIdx(69)]);
    set2bit(70,SIGN( + getDecValue(get2bit(23,Sc[getBlockIdx(23)])) + getDecValue(get2bit(68,Sc[getBlockIdx(68)]))), &S[getBlockIdx(70)]);
    set2bit(71,SIGN( - (getDecValue(get2bit(35,Sc[getBlockIdx(35)]))) - (getDecValue(get2bit(83,Sc[getBlockIdx(83)])))), &S[getBlockIdx(71)]);
    set2bit(72,SIGN( - (getDecValue(get2bit(1,Sc[getBlockIdx(1)]))) + getDecValue(get2bit(11,Sc[getBlockIdx(11)])) + getDecValue(get2bit(34,Sc[getBlockIdx(34)])) + getDecValue(get2bit(87,Sc[getBlockIdx(87)])) + 1),&S[getBlockIdx(72)]);
    set2bit(73,SIGN( - (getDecValue(get2bit(25,Sc[getBlockIdx(25)]))) + getDecValue(get2bit(43,Sc[getBlockIdx(43)])) - (getDecValue(get2bit(44,Sc[getBlockIdx(44)]))) + getDecValue(get2bit(45,Sc[getBlockIdx(45)])) - (getDecValue(get2bit(47,Sc[getBlockIdx(47)]))) + getDecValue(get2bit(66,Sc[getBlockIdx(66)])) + getDecValue(get2bit(67,Sc[getBlockIdx(67)]))),&S[getBlockIdx(73)]);
    set2bit(74,SIGN( + getDecValue(get2bit(73,Sc[getBlockIdx(73)]))), &S[getBlockIdx(74)]);
    set2bit(75,SIGN( + getDecValue(get2bit(66,Sc[getBlockIdx(66)])) + getDecValue(get2bit(74,Sc[getBlockIdx(74)]))),&S[getBlockIdx(75)]);
    set2bit(76,SIGN( - (getDecValue(get2bit(21,Sc[getBlockIdx(21)]))) - (getDecValue(get2bit(32,Sc[getBlockIdx(32)]))) - (getDecValue(get2bit(95,Sc[getBlockIdx(95)]))) + 3),&S[getBlockIdx(76)]);
    set2bit(77,SIGN( + getDecValue(get2bit(25,Sc[getBlockIdx(25)])) + getDecValue(get2bit(34,Sc[getBlockIdx(34)])) + getDecValue(get2bit(38,Sc[getBlockIdx(38)])) + (-1)),&S[getBlockIdx(77)]);
    set2bit(78,SIGN( + getDecValue(get2bit(12,Sc[getBlockIdx(12)])) + getDecValue(get2bit(21,Sc[getBlockIdx(21)])) + getDecValue(get2bit(25,Sc[getBlockIdx(25)])) + getDecValue(get2bit(34,Sc[getBlockIdx(34)])) + getDecValue(get2bit(38,Sc[getBlockIdx(38)])) - (getDecValue(get2bit(43,Sc[getBlockIdx(43)]))) - (getDecValue(get2bit(62,Sc[getBlockIdx(62)]))) - (getDecValue(get2bit(91,Sc[getBlockIdx(91)]))) + (-4)),&S[getBlockIdx(78)]);
    set2bit(79,SIGN( + getDecValue(get2bit(34,Sc[getBlockIdx(34)])) + getDecValue(get2bit(38,Sc[getBlockIdx(38)]))),&S[getBlockIdx(79)]);
    set2bit(80,SIGN( + getDecValue(get2bit(49,Sc[getBlockIdx(49)])) + getDecValue(get2bit(54,Sc[getBlockIdx(54)])) + (-1)),&S[getBlockIdx(80)]);
    set2bit(81,SIGN( + getDecValue(get2bit(55,Sc[getBlockIdx(55)])) + getDecValue(get2bit(57,Sc[getBlockIdx(57)])) + (-1)),&S[getBlockIdx(81)]);
    set2bit(82,SIGN( + getDecValue(get2bit(3,Sc[getBlockIdx(3)]))),&S[getBlockIdx(82)]);
    set2bit(83,SIGN( + getDecValue(get2bit(21,Sc[getBlockIdx(21)])) + getDecValue(get2bit(38,Sc[getBlockIdx(38)])) + getDecValue(get2bit(59, Sc[getBlockIdx(59)]))), &S[getBlockIdx(83)]);
    set2bit(84,SIGN( + getDecValue(get2bit(31,Sc[getBlockIdx(31)])) - (getDecValue(get2bit(82,Sc[getBlockIdx(82)])))), &S[getBlockIdx(84)]);
    set2bit(85,SIGN( - (getDecValue(get2bit(38,Sc[getBlockIdx(38)]))) + 1), &S[getBlockIdx(85)]);
    set2bit(86,SIGN( + getDecValue(get2bit(19,Sc[getBlockIdx(19)])) + getDecValue(get2bit(33,Sc[getBlockIdx(33)]))), &S[getBlockIdx(86)]);
    set2bit(87,SIGN( + getDecValue(get2bit(9,Sc[getBlockIdx(9)]))), &S[getBlockIdx(87)]);
    set2bit(88,SIGN( + getDecValue(get2bit(87,Sc[getBlockIdx(87)]))), &S[getBlockIdx(88)]);
    set2bit(89,SIGN( + getDecValue(get2bit(87,Sc[getBlockIdx(87)]))), &S[getBlockIdx(89)]);
    set2bit(90,SIGN( + getDecValue(get2bit(24,Sc[getBlockIdx(24)])) + getDecValue(get2bit(86,Sc[getBlockIdx(86)]))), &S[getBlockIdx(90)]);
    set2bit(91,SIGN( - (getDecValue(get2bit(90,Sc[getBlockIdx(90)]))) + 1), &S[getBlockIdx(91)]);
    set2bit(92,SIGN( + getDecValue(get2bit(7,Sc[getBlockIdx(7)])) + getDecValue(get2bit(43, Sc[getBlockIdx(43)])) + (-1)), &S[getBlockIdx(92)]);
    set2bit(93,SIGN( + getDecValue(get2bit(34,Sc[getBlockIdx(34)])) + getDecValue(get2bit(38, Sc[getBlockIdx(38)])) + (-1)), &S[getBlockIdx(93)]);
    set2bit(94,SIGN( + getDecValue(get2bit(93,Sc[getBlockIdx(93)]))), &S[getBlockIdx(94)]);
    set2bit(95,SIGN( + getDecValue(get2bit(21,Sc[getBlockIdx(21)])) - (getDecValue(get2bit(29, Sc[getBlockIdx(29)]))) - (getDecValue(get2bit(43,Sc[getBlockIdx(43)]))) + getDecValue(get2bit(61,Sc[getBlockIdx(61)])) + (-1)),&S[getBlockIdx(95)]);
}

__global__
void findAttractor(uint64 *attractors, uint32_t *transients, uint32_t *periods, uint32_t numThreads){
    int transient = 0, period = 0;
    uint64 S0[N], S1[N];
    uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t step =  NUM_STATES / NUM_COPYS; 
    uint32_t rest =  NUM_STATES % NUM_COPYS;
    uint32_t begin = 0;
    uint32_t end = step - 1;
    
    hiprandState_t state;
	hiprand_init(thread, /* the seed controls the sequence of random values that are produced */
              0, /* the sequence number is only important with multiple cores */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &state);
              
    bool flag = true;
    
    if(thread < numThreads){
        if(rest > 0){
            end = end + 1;
            rest = rest - 1;
        }else{
        	flag = false;
        }
        
        
        for(uint32_t i = 0; i < NUM_COPYS;i++){
            if(i == thread) break;
            
			if(rest > 0){
                end = end + 1;
                begin = begin + 1;
                rest = rest - 1;
            }else if(rest == 0 && flag){
                begin = begin + 1;
                flag = 0;
            }
            begin += step;
            end += step;
        }

	    for (int i = begin; i < end; ++i) {
	        S0[0] = S1[0] = thread;
	        int x = hiprand(&state) % 2;
	        int y = hiprand(&state) % 2;
	        int x_s = hiprand(&state) % 2;
	        int y_s = hiprand(&state) % 2;
	        S0[1] = S1[1] =  (x_s) ? x_s*x : x;// inicializar com rand
	        S0[2] = S1[2] = (y_s) ? y_s*y : y;// inicializar com rand
			set2bit(6, 1, &S0[getBlockIdx(6)]);      //Obrigatório **Conferir se esta setando o bit certo**
	    	set2bit(7, 1, &S1[getBlockIdx(7)]);      //Obrigatório **Conferir se esta setando o bit certo**
	        transient = 0;
	        period = 0;
	        do{
	            pass(S0);
	            pass(S0);
	            pass(S1);
	            transient++;
	        }while(!comp(S0,S1));
	
	        do{
	            pass(S0);
	            period++;
	        }while (!comp(S0,S1));
	        period--;
	        
	        transients[i] = transient;
	        periods[i]= period;
	        for(int s = 0; s < N; s++){
	            attractors[i * N + s] = S0[s];
	        }
	        //cout << transient << " " << period << std::endl;
	    }
	}	
}

int main() {
	// Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    // Print the vector length to be used, and compute its size
    size_t numNos = NUM_NOS;
    size_t numState = NUM_STATES;
    size_t size = numNos * numState * sizeof(bool);
    size_t size_transients = numState*sizeof(uint32_t);
    size_t size_periods = numState*sizeof(uint32_t);
    size_t totalBytes = size+size_transients+size_periods;
    size_t kb = totalBytes/(1024);
    size_t mb = kb/(1024);
    size_t gb = mb/(1024);
    printf("Find attractors net %lu nodes in %lu initials states.\n", numNos,numState);
    printf("Memory usage: %lu Gb or %lu Mb or %lu Kb.\n", gb, mb, kb);
    
	uint32_t *h_transients = (uint32_t*)malloc(size_transients);
    // Verifica se houve sucesso na aloca��o do vetor h_transients
    if (h_transients == NULL){
        fprintf(stderr, "Failed to allocate h_transients!\n");
        exit(EXIT_FAILURE);
    }    
    
    
    uint32_t *h_periods = (uint32_t*)malloc(size_periods);
    // Verifica se houve sucesso na aloca��o do vetor h_periods
    if (h_periods == NULL){
        fprintf(stderr, "Failed to allocate h_periods!\n");
        exit(EXIT_FAILURE);
    }
    
    
    //Aloca o vetor para a saida no host
    bool *h_attractors = (bool *)malloc(size);
    // Verifica se houve sucesso na aloca��o do vetor h_attractors
    if (h_attractors == NULL){
        fprintf(stderr, "Failed to allocate h_attractors!\n");
        exit(EXIT_FAILURE);
    }
    
    // Aloca os vetores na GPU (device)
    uint32_t *d_transients = NULL;
    err = hipMalloc((void **)&d_transients, NUM_STATES * sizeof(uint32_t));

    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate d_transients (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    uint32_t *d_periods = NULL;
    err = hipMalloc((void **)&d_periods, NUM_STATES * sizeof(uint32_t));

    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate d_periods (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    bool *d_attractors = NULL;
    err = hipMalloc((void **)&d_attractors, size);

    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate d_attractors (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    uint32_t threadsPerBlock = 256;
    uint32_t blocksPerGrid = (NUM_COPYS + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    
    findAttractor<<< blocksPerGrid, threadsPerBlock >>>(d_attractors, d_transients, d_periods, NUM_COPYS);
    
    //err = hipGetLastError();
    //if (err != hipSuccess){
    //    fprintf(stderr, "Failed to launch findAttractor kernel (error code %s)!\n", hipGetErrorString(err));
    //    exit(EXIT_FAILURE);
    //}

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    
	err = hipMemcpy(h_transients, d_transients, size_transients, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector d_transients from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(h_periods, d_periods, size_periods, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector d_periods from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(h_attractors, d_attractors, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector d_attractors from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    /*
    printf("Attractor found:\n");
    for(int i = 0; i < numState; i++){
       for(int j = 0; j < numNos; j++){
          printf("%d",h_attractors[getIndice(numNos,i,j)]);
      }
      printf("\n");
    }
    printf("\n");
    */
    err = hipFree(d_transients);

    if (err != hipSuccess){
        fprintf(stderr, "Failed to free device vector d_transients (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipFree(d_periods);

    if (err != hipSuccess){
        fprintf(stderr, "Failed to free device vector d_periods (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipFree(d_attractors);

    if (err != hipSuccess){
        fprintf(stderr, "Failed to free device vector d_attractors (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_transients);
    free(h_periods);
    free(h_attractors);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess){
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}
