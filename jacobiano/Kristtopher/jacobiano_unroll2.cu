#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#define DEBUG 0

void initialDataTriple(double *ip,double *ip2,double *ip3, unsigned long long int size){
    for(unsigned long long int i = 0; i < size; i++){
        ip3[i] = ip2[i] = ip[i] = (double)i;
        //ip3[i] = ip2[i] = ip[i] = (double)(rand() & 0xFF ) / 10.0f;
    }
    return;
}

void imprime(double *ip, unsigned long long int size, int nx){
    for(unsigned long long int i = 0; i < size; i++){
        if(i % nx==0)
            printf("\n");
        printf("%.4f\t",ip[i]);
    }
    printf("\n");
    return;
}

void checkResult(double *hostRef, double *gpuRef, unsigned long long int N){
    double epsilon = 1.0E-8;
    //epsilon = 0.001; //3.9;       // new error
    bool match = 1;
    for (unsigned long long int i = 0; i < N; i++){
        if (abs(hostRef[i] - gpuRef[i]) > epsilon){
            match = 0;
            printf("host %f gpu %f index %llu\n", hostRef[i], gpuRef[i],i);
            break;
        }
    }
    if (match)
        printf("Arrays match.\n\n");
    else
        printf("Arrays do not match.\n\n");
}

void jacobiOnHost(int iter_max, double *matriz, double* host_ref, int dim_x, int dim_y){
    double *ia = matriz;
    double *Anew = host_ref;
    for (int k = 0; k < iter_max; ++k) {
        ia = matriz + dim_y;
        Anew = host_ref + dim_y;
         for( int i = 1; i < dim_x-1; i++) {
            for(int j = 1; j < dim_y-1; j++) {
              Anew[j] = 0.25 * (ia[j+1] + ia[j-1] + ia[j-dim_y] + ia[j+dim_y]);
            }
            ia += dim_y;
            Anew += dim_y;
        }
        ia = matriz;
        Anew = host_ref;
        for(int i = 0; i < (dim_x * dim_y); i++)
            ia[i] = Anew[i];
    }
    return;
}

// grid 2D block 2D
__global__ void jacobiOnGpuNaive2D(double *MatA, double *d_MatC, int dim_x, int dim_y,int iter_max){
    unsigned long long int i = blockDim.x * blockIdx.x + threadIdx.x; 
    unsigned long long int j = blockDim.y * blockIdx.y + threadIdx.y *2; 
    unsigned long long int iPrev = i-1; 
    unsigned long long int iNext = i+1; 
    unsigned long long int jPrev = j-1; 
    unsigned long long int jNext = j+1; 
    unsigned long long int index = i * dim_y + j;
    unsigned long long int indexUnroll2 = index - 1;
    for (int k = 0; k < iter_max; k++){
        if( i > 0 && j > 0 && i < (dim_x-1) && j <(dim_y-1)){
            d_MatC[index] = 0.25f * (MatA[iPrev * dim_y + j] + MatA[iNext* dim_y + j] + MatA[i * dim_y+ jPrev] + MatA[i* dim_y + jNext]);
            d_MatC[indexUnroll2] = 0.25f * (MatA[indexUnroll2 + 1] + MatA[indexUnroll2 - 1] + MatA[indexUnroll2 + dim_x] + MatA[indexUnroll2 - dim_x]);
            if(DEBUG){
                printf("index %llu \n", index);
                printf("indexUnroll2 %llu, contornos %lf %lf %lf %lf \n",indexUnroll2, MatA[indexUnroll2 + 1], MatA[indexUnroll2 - 1], MatA[indexUnroll2 + dim_y] , MatA[indexUnroll2 - dim_y] );
            }
        }
       __syncthreads();
       if(index<dim_x*dim_y) MatA[index] = d_MatC[index];
    }   
}

int main(int argc, char **argv){
    printf("%s Starting...\n", argv[0]);
    // set up data size of matrix
    int iter_max = 2;//20
    int nx = 5700;
    int ny = 5700;
    int threadsBlocoX = 32;// x*y <= 1024
    int threadsBlocoY = 32;

    if (argc > 1) ny = nx = atoi(argv[1]);
    if (argc > 2) threadsBlocoX = atoi(argv[2]);
    if (argc > 3) threadsBlocoY = atoi(argv[3]);

    unsigned long long int nxy = nx * ny;
    unsigned long long int nBytes = nxy * sizeof(double);
    printf("Matrix size: nx %d ny %d total size %llu\n", nx, ny, nxy);

    // malloc host memory
    double *h_Matriz,*hostRef, *gpuRef;
    h_Matriz = (double*)malloc(nBytes);
    hostRef = (double *)malloc(nBytes);
    gpuRef = (double *)malloc(nBytes);

    // initialize data at host side
    double iStart = seconds();
    initialDataTriple(h_Matriz,hostRef,gpuRef, nxy);
    double iElaps = seconds() - iStart;
    printf("initialize matrix elapsed %f sec\n", iElaps);
    if(DEBUG)
        imprime(h_Matriz, nxy,nx);

    // Jacobi at host side for result checks
    iStart = seconds();
    jacobiOnHost(iter_max, h_Matriz, hostRef, ny, nx);
    iElaps = seconds() - iStart;
    printf("jacobiOnHost elapsed %f sec\n", iElaps);
    if(DEBUG)
        imprime(hostRef, nxy,nx);
    
    // malloc device global memory
    double *d_MatA, *d_MatC;
    CHECK(hipMalloc((void **)&d_MatA, nBytes));
    CHECK(hipMalloc((void **)&d_MatC, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_MatA, gpuRef, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_MatC, gpuRef, nBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side 2D-2D
    dim3 dimBlock(threadsBlocoX, threadsBlocoY);
    //dim3 dimGrid(1,1);// /2
    dim3 dimGrid(nx/dimBlock.x/2,ny/dimBlock.y/2);// /2
    iStart = seconds();
    jacobiOnGpuNaive2D<<<dimGrid, dimBlock>>>(d_MatA, d_MatC, nx, ny,iter_max);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("sumMatrixOnGPU2D <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", dimGrid.x,dimGrid.y,dimBlock.x, dimBlock.y, iElaps);
    
    // check kernel error
    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost));
    if(DEBUG)
        imprime(gpuRef, nxy,nx);
    
    // check device results
    checkResult(hostRef, gpuRef, nxy);

    // free device global memory
    CHECK(hipFree(d_MatA));
    CHECK(hipFree(d_MatC));

    // free host memory
    free(h_Matriz);
    free(hostRef);
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());
    return (0);
}