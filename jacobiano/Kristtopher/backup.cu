#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This example demonstrates a simple vector sum on the GPU and on the host.
 * sumArraysOnGPU splits the work of the vector sum across CUDA threads on the
 * GPU. A 1D thread block and 1D grid are used. sumArraysOnHost sequentially
 * iterates through vector elements on the host.
 */

void initialDataSingle(float *ip, const int size){
    int i;
    for(i = 0; i < size; i++){
        //ip[i] = i;
	ip[i] = (float)(rand() & 0xFF ) / 10.0f;
    }
    return;
}
void initialDataDouble(float *ip,float *ip2, const int size){
    int i;
    for(i = 0; i < size; i++){
        //ip[i] = i;
	ip2[i] = ip[i] = (float)(rand() & 0xFF ) / 10.0f;
    }
    return;
}
void initialDataTriple(float *ip,float *ip2,float *ip3, const int size){
    for(int i = 0; i < size; i++){
        //ip[i] = i;
	ip3[i] = ip2[i] = ip[i] = (float)(rand() & 0xFF ) / 10.0f;
    }
    return;
}

void initialDataEdge(float *ip,float *ip2,float *ip3, int nx, int ny){
    memset(ip,  0, nx * ny * sizeof(float));
	memset(ip2, 0, nx * ny * sizeof(float));
	memset(ip3, 0, nx * ny * sizeof(float));
	for(int i = 0; i < (nx * ny); i++){
		if(i < nx || i > (nx*ny-nx))
	        ip3[i] = ip2[i] = ip[i] = (float)(rand() & 0xFF ) / 10.0f;
		else if(i% nx == 0){
		    ip3[i] = ip2[i] = ip[i] = (float)(rand() & 0xFF ) / 10.0f;
			ip3[i+nx-1] = ip2[i+nx-1] = ip[i+nx-1] = (float)(rand() & 0xFF ) / 10.0f;
		}
    }
    return;
}
void imprime(float *ip, const int size, int nx){
    int i;
    for(i = 0; i < size; i++){
	if(i%nx==0)
	    printf("\n");
        printf("%.4f\t",ip[i]);
    }
    printf("\n");
    return;
}
void jacobiOnHost(int iter_max, float *matriz, float* host_ref, int dim_x, int dim_y){
    int iter = 0;
    float *ia = matriz;
    float *Anew = host_ref;
    
    for (int k = 0; k < iter_max; ++k) {
	ia = matriz + dim_y;
    	Anew = host_ref + dim_y;
         for( int i = 1; i < dim_x-1; i++) {
            for(int j = 1; j < dim_y-1; j++) {	
		printf("%.4f, %.4f, %.4f, %.4f\n",ia[j+1], ia[j-1], ia[j-dim_y], ia[j+dim_y]);
		Anew[j] = 0.25 * (ia[j+1] + ia[j-1] + ia[j-dim_y] + ia[j+dim_y]);
	    }
	    ia += dim_y;
            Anew += dim_y;
        }
	ia = matriz;
    	Anew = host_ref;
	for(int i = 0; i < (dim_x * dim_y); i++){
            ia[i] = Anew[i];
    	}
        iter++;
    }
	return;
}

// grid 1D block 1D

__global__ void jacobiOnGpuNaive(int iter_max, float *MatA, float *MatC, int dim_x, int dim_y){
    //unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iter = 0;
    float *ia = MatA;
    float *Anew = MatC; 
    for (int k = 0; k < iter_max; ++k) {
	ia = MatA + dim_y;
    	Anew = MatC + dim_y;
         for( int i = 1; i < dim_x-1; i++) {
            for(int j = 1; j < dim_y-1; j++) {	
		printf("%.4f, %.4f, %.4f, %.4f\n",ia[j+1], ia[j-1], ia[j-dim_y], ia[j+dim_y]);
		Anew[j] = 0.25 * (ia[j+1] + ia[j-1] + ia[j-dim_y] + ia[j+dim_y]);
	    }
	    ia += dim_y;
            Anew += dim_y;
        }
	ia = MatA;
    	Anew = MatC;
	for(int i = 0; i < (dim_x * dim_y); i++){
            ia[i] = Anew[i];
    	}
        iter++;
    }
}

void checkResult(float *hostRef, float *gpuRef, const int N){
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++){
        if (abs(hostRef[i] - gpuRef[i]) > epsilon){
            match = 0;
            printf("host %f gpu %f\n", hostRef[i], gpuRef[i]);
            break;
        }
    }

    if (match)
        printf("Arrays match.\n\n");
    else
        printf("Arrays do not match.\n\n");
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of matrix
    int iter_max =1;
    int nx = 3;//1 << 14;
    int ny = 5;//1 << 14;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);
    printf("Matrix size: nx %d ny %d\n", nx, ny);

    // malloc host memory
    float *h_Matriz,*hostRef, *gpuRef;
    h_Matriz = (float*)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    // initialize data at host side
    double iStart = seconds();
    //initialDataEdge(h_Matriz,hostRef,gpuRef, nx, ny);
	//imprime(h_Matriz, nxy,nx);
	initialDataTriple(h_Matriz,hostRef,gpuRef, nxy);
    double iElaps = seconds() - iStart;
    printf("initialize matrix elapsed %f sec\n", iElaps);
    //imprime(h_Matriz, nxy,nx);

    // Jacobi at host side for result checks
    iStart = seconds();
    jacobiOnHost(iter_max, h_Matriz, hostRef, ny, nx);
    iElaps = seconds() - iStart;
    printf("jacobiOnHost elapsed %f sec\n", iElaps);
    imprime(hostRef, nxy,nx);
    
    // malloc device global memory
    float *d_MatA, *d_MatC;
    CHECK(hipMalloc((void **)&d_MatA, nBytes));
    CHECK(hipMalloc((void **)&d_MatC, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_MatA, gpuRef, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_MatC, gpuRef, nBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    int dimx = 32;//32
    dim3 block(dimx, 1);
    dim3 grid((nx + block.x - 1) / block.x, 1);

    iStart = seconds();
    jacobiOnGpuNaive<<<grid, block>>>(iter_max, d_MatA, d_MatC, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("sumMatrixOnGPU1D <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", grid.x,
           grid.y,
           block.x, block.y, iElaps);

    // check kernel error
    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost));
    imprime(gpuRef, nxy,nx);
    // check device results
    checkResult(hostRef, gpuRef, nxy);

    // free device global memory
    CHECK(hipFree(d_MatA));
    CHECK(hipFree(d_MatC));

    // free host memory
    free(h_Matriz);
    free(hostRef);
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());
    return (0);
}
