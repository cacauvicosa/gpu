#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#define DIM 128


void initialDataTriple(double *ip,double *ip2,double *ip3, unsigned long long int size){
    for(unsigned long long int i = 0; i < size; i++){
        ip3[i] = ip2[i] = ip[i] = (double)i;
        //ip3[i] = ip2[i] = ip[i] = (double)(rand() & 0xFF ) / 10.0f;
    }
    return;
}

void checkResult(double *hostRef, double *gpuRef, unsigned long long int N,double epsilon){
    bool match = 1;
    epsilon = 0.000001;
    for (unsigned long long int i = 0; i < N; i++){
        if (abs(hostRef[i] - gpuRef[i]) > epsilon){
            match = 0;
            printf("host %f gpu %f index %llu\n", hostRef[i], gpuRef[i],i);
            break;
        }
    }
    if (match)
        printf("Arrays match.\n\n");
    else
        printf("Arrays do not match.\n\n");
}

void jacobiOnHost(int iter_max, double *matriz, double* host_ref, int dim_x, int dim_y){
    double *ia = matriz;
    double *Anew = host_ref;
    for (int k = 0; k < iter_max; ++k) {
        ia = matriz + dim_y;
        Anew = host_ref + dim_y;
         for( int i = 1; i < dim_x-1; i++) {
            for(int j = 1; j < dim_y-1; j++) {
                Anew[j] = 0.25 * (ia[j+1] + ia[j-1] + ia[j-dim_y] + ia[j+dim_y]);
            }
            ia += dim_y;
            Anew += dim_y;
        }
        ia = matriz;
        Anew = host_ref;
        for(int i = 0; i < (dim_x * dim_y); i++)
            ia[i] = Anew[i];
    }
    return;
}

// grid 2D block 2D-2               Dd_MatA,         d_MatC,        nx,         ny,     iter_max, epsilon,      d_odata,    d_idata
__global__ void jacobiOnGpuNaive2D(double *MatA, double *d_MatC, int dim_x, int dim_y,int iter_max, double tol, double *g_odata, double *idata){
    int  k = 0;
    unsigned long long int i = blockDim.x * blockIdx.x + threadIdx.x; 
    unsigned long long int j = blockDim.y * blockIdx.y + threadIdx.y; 
    unsigned long long int iPrev = i-1; 
    unsigned long long int iNext = i+1; 
    unsigned long long int jPrev = j-1; 
    unsigned long long int jNext = j+1; 
    unsigned long long int index = i * dim_y + j;
    //for (int k = 0; k < iter_max; k++){
    do{
        if( i > 0 && j > 0 && i < (dim_x-1) && j <(dim_y-1)){
            d_MatC[index] = 0.25f * (MatA[iPrev * dim_y + j] + MatA[iNext* dim_y + j] + MatA[i * dim_y+ jPrev] + MatA[i* dim_y + jNext]);
        }
        __syncthreads();
        if(index<dim_x*dim_y) {
            idata[index] = abs (MatA[index] - d_MatC[index]);
            MatA[index] = d_MatC[index];
        }
        __syncthreads();
        //printf("%d\n",k);
        for (int stride = 1; stride < blockDim.x; stride *= 2){
            if ((index % (2 * stride)) == 0){
            idata[index] += idata[index + stride];
        }
        __syncthreads();
    }

    if (index == 0) 
        g_odata[blockIdx.x] = idata[0];
    }while(g_odata[blockIdx.x] > tol && ++k < iter_max);  
    //printf("reducção %f",idata[0]);
}

int main(int argc, char **argv){
    printf("%s Starting...\n", argv[0]);
    double epsilon = 1.0E-8;
    epsilon = 1.5;
    // set up data size of matrix
    int iter_max = 20;
    int nx = 5700;
    int ny = 5700;
    int threadsBlocoX = 32; // x*y <= 1024
    int threadsBlocoY = 32;

    if (argc > 1) ny = nx = atoi(argv[1]);
    if (argc > 2) threadsBlocoX = atoi(argv[2]);
    if (argc > 3) threadsBlocoY = atoi(argv[3]);
    
    unsigned long long int nxy = nx * ny;
    unsigned long long int nBytes = nxy * sizeof(double);
    printf("Matrix size: nx %d ny %d total size %llu\n", nx, ny, nxy);

    // malloc host memory
    double *h_Matriz,*hostRef, *gpuRef;
    h_Matriz = (double*)malloc(nBytes);
    hostRef = (double *)malloc(nBytes);
    gpuRef = (double *)malloc(nBytes);
    double *d_odata = NULL;
    size_t bytes = nxy * sizeof(double);
    double *h_idata = (double *) malloc(bytes);

    // initialize data at host side
    double iStart = seconds();
    initialDataTriple(h_Matriz,hostRef,gpuRef, nxy);
    double iElaps = seconds() - iStart;
    printf("initialize matrix elapsed %f sec\n", iElaps);

    // Jacobi at host side for result checks
    iStart = seconds();
    jacobiOnHost(iter_max, h_Matriz, hostRef, ny, nx);
    iElaps = seconds() - iStart;
    printf("jacobiOnHost elapsed %f sec\n", iElaps);
    
    // malloc device global memory
    double *d_MatA, *d_MatC;
    double *d_idata = NULL;
    CHECK(hipMalloc((void **)&d_MatA, nBytes));
    CHECK(hipMalloc((void **)&d_MatC, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_MatA, gpuRef, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_MatC, gpuRef, nBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side 2D-2D
    dim3 dimBlock(threadsBlocoX, threadsBlocoY);
    dim3 dimGrid(nx/dimBlock.x,ny/dimBlock.y);
    double *h_odata = (double *) malloc(dimGrid.x * sizeof(double));
    CHECK(hipMalloc((void **) &d_odata, dimGrid.x * sizeof(double)));
    CHECK(hipMalloc((void **) &d_idata, bytes));
    iStart = seconds();
    jacobiOnGpuNaive2D<<<dimGrid, dimBlock>>>(d_MatA, d_MatC, nx, ny,iter_max, epsilon, d_odata, d_idata);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(h_odata, d_odata, dimGrid.x * sizeof(double),hipMemcpyDeviceToHost));

    double gpu_sum = 0.0;
    for (int i = 0; i < dimGrid.x; i++) gpu_sum += h_odata[i];

    printf("gpu Neighbored gpu_sum: %lf \n", gpu_sum);
    iElaps = seconds() - iStart;
    printf("sumMatrixOnGPU2D <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", dimGrid.x,dimGrid.y,dimBlock.x, dimBlock.y, iElaps);

    // check kernel error
    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost));
    
    // check device results
    checkResult(hostRef, gpuRef, nxy,epsilon);

    // free device global memory
    CHECK(hipFree(d_MatA));
    CHECK(hipFree(d_MatC));

    // free host memory
    free(h_Matriz);
    free(hostRef);
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());
    return (0);
}